#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <time.h>
#include <vector>
#include <opencv2/opencv.hpp>
#include "opencv2/highgui.hpp"
#include <iostream>
#include <math.h>
#include <chrono>

inline __device__
float __char_as_float(uchar b8)
{
    return __uint2float_rn(b8) / 127.5f - 1.f;
}

inline __device__
uchar __float2uchar(float f16)
{
    return __float2uint_rd(f16);
    // return __float2uint_rd(f16 * 255.f);
}

static inline void _safe_cuda_call(hipError_t err, const char* msg, const char* file_name, const int line_number)
{
    if (err != hipSuccess)
    {
        fprintf(stderr, "%s\n\nFile: %s\n\nLine Number: %d\n\nReason: %s\n", msg, file_name, line_number, hipGetErrorString(err));
        std::cin.get();
        exit(EXIT_FAILURE);
    }
}

#define SAFE_CALL(call,msg) _safe_cuda_call((call),(msg),__FILE__,__LINE__)

inline __device__
int clip(int x, int a, int b)
{
    return x >= a ? (x < b ? x : b - 1) : a;
}

// enlarge the original image k times in x and y direction
// write dataOut coalesced
__global__ void inter_nearest_k(uchar3* dataIn, uchar3* dataOut, int imgHeight, int imgWidth, int imgHeight_k, int imgWidth_k, int k)
{
    int xIdx = threadIdx.x + blockIdx.x * blockDim.x;
    int yIdx = threadIdx.y + blockIdx.y * blockDim.y;

    if (xIdx < imgWidth_k && yIdx < imgHeight_k)
    {
        dataOut[yIdx * imgWidth_k + xIdx] = dataIn[(yIdx / k) * imgWidth + xIdx / k];
    }
}

// bilinear interpolation, enlarge k times in x and y direction
// 浮点型乘法
__global__ void inter_liner_k(uchar3* dataIn, uchar3* dataOut, int imgHeight, int imgWidth, int imgHeight_k, int imgWidth_k, float scale)
{
    // __shared__ float shared_eles[34][34];
    int xIdx = threadIdx.x + blockIdx.x * blockDim.x;
    int yIdx = threadIdx.y + blockIdx.y * blockDim.y;

    if (xIdx < imgWidth_k && yIdx < imgHeight_k)
    {
        float fx = (float)((xIdx + 0.5f) * scale - 0.5f);
        int sx = floorf(fx);
        fx -= sx;
        sx = min(sx, imgWidth - 1);
        int sx2 = min(sx + 1, imgWidth - 1);
        if (sx < 0)
            sx2 = 0, sx = 0;

        float2 cbufx;
        cbufx.x = 1.f - fx;
        cbufx.y = fx;

        float fy = (float)((yIdx + 0.5f) * scale - 0.5f);
        int sy = floorf(fy);
        fy -= sy;
        sy = min(sy, imgHeight - 1);
        int sy2 = min(sy + 1, imgHeight - 1);
        if (sy < 0)
            sy2 = 0, sy = 0;

        float2 cbufy;
        cbufy.x = 1.f - fy;
        cbufy.y = fy;

        // if(sx % 2 == 0 || sy % 2 == 0)
        // uchar3 s11 = make_uchar3(0,0,0), s12 = make_uchar3(0,0,0), s21 = make_uchar3(0,0,0), s22 = make_uchar3(0,0,0);
        // 从global memory加载数据花费1500 us
        uchar3 s11 = dataIn[sy * imgWidth + sx];
        uchar3 s12 = dataIn[sy * imgWidth + sx2];
        uchar3 s21 = dataIn[sy2 * imgWidth + sx];
        uchar3 s22 = dataIn[sy2 * imgWidth + sx2];
        // __syncthreads();

        float h_rst00x, h_rst01x, h_rst00y, h_rst01y, h_rst00z, h_rst01z;
        h_rst00x = s11.x * cbufx.x + s12.x * cbufx.y;
        h_rst01x = s21.x * cbufx.x + s22.x * cbufx.y;
        h_rst00y = s11.y * cbufx.x + s12.y * cbufx.y;
        h_rst01y = s21.y * cbufx.x + s22.y * cbufx.y;
        h_rst00z = s11.z * cbufx.x + s12.z * cbufx.y;
        h_rst01z = s21.z * cbufx.x + s22.z * cbufx.y;

        // 写入global memory花费1500 us， 所有其他的运算花费1000 us
        dataOut[yIdx * imgWidth_k + xIdx].x = h_rst00x * cbufy.x + h_rst01x * cbufy.y; // B
        dataOut[yIdx * imgWidth_k + xIdx].y = h_rst00y * cbufy.x + h_rst01y * cbufy.y; // G
        dataOut[yIdx * imgWidth_k + xIdx].z = h_rst00z * cbufy.x + h_rst01z * cbufy.y; // R
    }
}

int resize(void)
{
    int k = 11;
    float scale = 1.f / (float)k;
    cv::Mat img_ori = cv::imread("lisfan-70.jpg");
    int imgWidth = img_ori.cols;
    int imgHeight = img_ori.rows;
    int imgHeight_k = imgHeight * k;
    int imgWidth_k = imgWidth * k;

    float runtime;
    

    uchar3* d_in;
    uchar3* d_out;

    cv::Mat img_resize_gpu(imgHeight_k, imgWidth_k, CV_8UC3);

    SAFE_CALL(hipMalloc((void**)&d_in, imgHeight * imgWidth * sizeof(uchar3)), "hipMalloc d_in fialed");
    SAFE_CALL(hipMalloc((void**)&d_out, imgHeight_k * imgWidth_k * sizeof(uchar3)), "hipMalloc d_out failed");

    SAFE_CALL(hipMemcpy(d_in, img_ori.data, imgHeight * imgWidth * sizeof(uchar3), hipMemcpyHostToDevice), "d_in hipMemcpyHostToDevice failed");

    dim3 threadsPerBlock(32, 32);
    dim3 blocksPerGrid((imgWidth_k + threadsPerBlock.x - 1) / threadsPerBlock.x, (imgHeight_k + threadsPerBlock.y - 1) / threadsPerBlock.y);

    inter_liner_k << <blocksPerGrid, threadsPerBlock >> > (d_in, d_out, imgHeight, imgWidth, imgHeight_k, imgWidth_k, scale);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    // hipDeviceSynchronize(); //CPU端计时，需要同步CPU和gpu，否则测速结果为cpu启动内核函数的速度
    // k=1, 562.528 us; k=3, 791.072 us; k=5, 1267.71 us; k=7, 1933.28 us; k=9, 2896.38 us; k=11, 3989.31 us
    // 不写入global memory，resize完了之后直接用
    // k=1, 525.568 us; k=3, 590.449 us; k=5, 754.656 us; k=7, 768.704 us; k=9, 985.76 us; k=11, 1086.08 us
    std::cout << "hipEvent_t time: " << runtime * 1000 << " us" << std::endl;

    SAFE_CALL(hipMemcpy(img_resize_gpu.data, d_out, imgHeight_k * imgWidth_k * sizeof(uchar3), hipMemcpyDeviceToHost), "d_out hipMemcpyDeviceToHost failed");

    cv::Mat resid(imgHeight_k, imgWidth_k, CV_8UC3);
    int diff = 0;
    for (int j = 0; j < imgHeight_k; j++)
    {
        for (int i = 0; i < imgWidth_k; i++)
        {
            resid.at<cv::Vec3b>(j, i)[0] = 100 * (img_resize_cpu.at<cv::Vec3b>(j, i)[0] - img_resize_gpu.at<cv::Vec3b>(j, i)[0]);
            resid.at<cv::Vec3b>(j, i)[1] = 100 * (img_resize_cpu.at<cv::Vec3b>(j, i)[1] - img_resize_gpu.at<cv::Vec3b>(j, i)[1]);
            resid.at<cv::Vec3b>(j, i)[2] = 100 * (img_resize_cpu.at<cv::Vec3b>(j, i)[2] - img_resize_gpu.at<cv::Vec3b>(j, i)[2]);
            if (abs(img_resize_cpu.at<cv::Vec3b>(j, i)[0] - img_resize_gpu.at<cv::Vec3b>(j, i)[0]) > 0 ||
                abs(img_resize_cpu.at<cv::Vec3b>(j, i)[1] - img_resize_gpu.at<cv::Vec3b>(j, i)[1]) > 0 ||
                abs(img_resize_cpu.at<cv::Vec3b>(j, i)[2] - img_resize_gpu.at<cv::Vec3b>(j, i)[2]) > 0) // > 1  diff: 0
            {
                diff += 1;
            }
        }
    }
    std::cout << "diff: " << diff << std::endl;

    SAFE_CALL(hipFree(d_in), "free d_in failed");
    SAFE_CALL(hipFree(d_out), "free d_out failed");

    cv::imwrite("lisfan-70_" + std::to_string(k) + "_gpu.jpg", img_resize_gpu);
    cv::imwrite("cpu_gpu_residual_" + std::to_string(k) + ".jpg", resid);

    return 0;
}
