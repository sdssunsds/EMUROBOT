#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <hip/device_functions.h>
#include <opencv2\opencv.hpp>
#include <iostream>
using namespace std;
__global__ void matSum(uchar* dataIn, int* dataOutSum, int* dataOutMax, int* dataOutMin, int imgHeight, int imgWidth)
{
    //__shared__ int _data[1600];
    const int number = 2048;

    extern __shared__ int _sum[];  //Сͼ�������͹�������
    __shared__ int _max[number];  //Сͼ����������ֵ��������
    __shared__ int _min[number];  //Сͼ���������Сֵ��������

    int thread = threadIdx.x + threadIdx.y * blockDim.x; //һ��block������thread������ֵ
    int threadIndex = threadIdx.x + threadIdx.y * imgWidth; //ÿ��С���д�����ݵ�thread����ֵ
    //ÿ��С���д�����ݵ�block����ֵ
    int blockIndex1 = blockIdx.x * blockDim.x + 2 * blockIdx.y * blockDim.y * imgWidth; //40*20���ϰ�block����ֵ
    int blockIndex2 = blockIdx.x * blockDim.x + (2 * blockIdx.y + 1) * blockDim.y * imgWidth; //40*20���°�block����ֵ

    int index1 = threadIndex + blockIndex1; //ÿ��block���ϰ벿������ֵ
    int index2 = threadIndex + blockIndex2; //ÿ��block���°벿������ֵ

    //���������40*40Сͼ����е���������ֵ�����δ��͵�����������
    _sum[thread] = dataIn[index1]; //���ϰ벿�ֵ�40*20���������ݸ�ֵ������������
    _sum[thread + blockDim.x * blockDim.y] = dataIn[index2]; //���°벿�ֵ�40*20���������ݸ�ֵ������������

    _max[thread] = dataIn[index1];
    _max[thread + blockDim.x * blockDim.y] = dataIn[index2];

    _min[thread] = dataIn[index1];
    _min[thread + blockDim.x * blockDim.y] = dataIn[index2];

    //memcpy(_sum, _data, 1600 * sizeof(int));
    //memcpy(_max, _data, 1600 * sizeof(int));
    //memcpy(_min, _data, 1600 * sizeof(int));  ��GPU��Device������memcpy�������п����ᵼ���Կ����ң��ʲ�ѡ����ַ�ʽ

    //���ù�Լ�㷨���40*40Сͼ�����1600������ֵ�еĺ͡����ֵ�Լ���Сֵ
    for (unsigned int s = number / 2; s > 0; s >>= 1)
    {
        if (thread < s)
        {
            _sum[thread] += _sum[thread + s];
            if (_max[thread] < _max[thread + s]) { _max[thread] = _max[thread + s]; }
            if (_min[thread] > _min[thread + s]) { _min[thread] = _min[thread + s]; }
        }
        __syncthreads(); //�����߳�ͬ��
    }
    if (threadIndex == 0)
    {
        //��ÿ��С���еĽ�����浽�����
        dataOutSum[blockIdx.x + blockIdx.y * gridDim.x] = _sum[0];
        dataOutMax[blockIdx.x + blockIdx.y * gridDim.x] = _max[0];
        dataOutMin[blockIdx.x + blockIdx.y * gridDim.x] = _min[0];
    }

}
